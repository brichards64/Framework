#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <vector>
#include <sys/time.h>
#include "hip/hip_runtime_api.h"
#include "library_daq.h"


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <thrust/extrema.h>

/////////////////////////////
// define global variables //
/////////////////////////////
/// parameters
double distance_between_vertices; // linear distance between test vertices
double wall_like_distance; // distance from wall (in units of distance_between_vertices) to define wall-like events
unsigned int time_step_size; // time binning for the trigger
__constant__ unsigned int constant_time_step_size; 
unsigned int water_like_threshold_number_of_pmts; // number of pmts above which a trigger is possible for water-like events
unsigned int wall_like_threshold_number_of_pmts; // number of pmts above which a trigger is possible for wall-like events
double coalesce_time; // time such that if two triggers are closer than this they are coalesced into a single trigger
double trigger_gate_up; // duration to be saved after the trigger time
double trigger_gate_down; // duration to be saved before the trigger time
/// detector
double detector_height; // detector height
double detector_radius; // detector radius
/// pmts
unsigned int n_PMTs; // number of pmts in the detector
__constant__ unsigned int constant_n_PMTs;
double * PMT_x, *PMT_y, *PMT_z; // coordinates of the pmts in the detector
/// vertices
unsigned int n_test_vertices; // number of test vertices
unsigned int n_water_like_test_vertices; // number of test vertices
__constant__ unsigned int constant_n_test_vertices;
__constant__ unsigned int constant_n_water_like_test_vertices;
double * vertex_x, * vertex_y, * vertex_z; // coordinates of test vertices
/// threads
unsigned int number_of_kernel_blocks;  // number of cores to be used
dim3 number_of_kernel_blocks_3d;
unsigned int number_of_threads_per_block; // number of threads per core to be used
dim3 number_of_threads_per_block_3d;
unsigned int grid_size;  // grid = (n cores) X (n threads / core)
/// hits
double time_offset;  // ns, offset to make times positive
__constant__ double constant_time_offset;
unsigned int n_time_bins; // number of time bins 
__constant__ unsigned int constant_n_time_bins;
unsigned int n_hits; // number of input hits from the detector
__constant__ unsigned int constant_n_hits;
unsigned int * host_ids; // pmt id of a hit
unsigned int *device_ids;
texture<unsigned int, 1, hipReadModeElementType> tex_ids;
unsigned int * host_times;  // time of a hit
unsigned int *device_times;
texture<unsigned int, 1, hipReadModeElementType> tex_times;
// npmts per time bin
unsigned int * device_n_pmts_per_time_bin; // number of active pmt in a time bin
// tof
double speed_light_water;
float *device_times_of_flight; // time of flight between a vertex and a pmt
float *host_times_of_flight;
texture<float, 1, hipReadModeElementType> tex_times_of_flight;
// triggers
std::vector<std::pair<unsigned int,unsigned int> > candidate_trigger_pair_vertex_time;  // pair = (v, t) = (a vertex, a time at the end of the 2nd of two coalesced bins)
std::vector<unsigned int> candidate_trigger_npmts_in_time_bin; // npmts in time bin
std::vector<std::pair<unsigned int,unsigned int> > trigger_pair_vertex_time;
std::vector<unsigned int> trigger_npmts_in_time_bin;
std::vector<std::pair<unsigned int,unsigned int> > final_trigger_pair_vertex_time;
std::vector<double> output_trigger_information;
// C timing
struct timeval t0;
struct timeval t1;
// CUDA timing
hipEvent_t start, stop, total_start, total_stop;
// find candidates
unsigned int * host_max_number_of_pmts_in_time_bin;
unsigned int * device_max_number_of_pmts_in_time_bin;
unsigned int *  host_vertex_with_max_n_pmts;
unsigned int *  device_vertex_with_max_n_pmts;
// gpu properties
int max_n_threads_per_block;
int max_n_blocks;
// verbosity level
bool use_verbose;
// files
std::string detector_file;
std::string pmts_file;

float elapsed_parameters, elapsed_pmts, elapsed_detector, elapsed_vertices,
  elapsed_threads, elapsed_tof, elapsed_memory_tofs_dev, elapsed_memory_candidates_host, elapsed_tofs_copy_dev,
  elapsed_input, elapsed_memory_dev, elapsed_copy_dev, elapsed_kernel, 
  elapsed_threads_candidates, elapsed_candidates_memory_dev, elapsed_candidates_kernel,
  elapsed_candidates_copy_host, choose_candidates, elapsed_coalesce, elapsed_gates, elapsed_free, elapsed_total,
  elapsed_tofs_free, elapsed_reset;
bool use_timing;

__global__ void kernel_correct_times(unsigned int *ct);
__global__ void kernel_find_vertex_with_max_npmts_in_timebin(unsigned int * np, unsigned int * mnp, unsigned int * vmnp);
__device__ unsigned int device_get_distance_index(unsigned int pmt_id, unsigned int vertex_block);
__device__ unsigned int device_get_time_index(unsigned int hit_index, unsigned int vertex_block);




//
// main code
//

int CUDAFunction(std::vector<int> PMTids, std::vector<int> times)
{


  ////////////////
  // read input //
  ////////////////
  // set: n_hits, host_ids, host_times, time_offset, n_time_bins
  // use: time_offset, n_test_vertices
  // memcpy: constant_n_time_bins, constant_n_hits
  if( use_timing )
    start_c_clock();
  if( !read_the_input(PMTids, times) ) return 0;
  if( use_timing )
    elapsed_input += stop_c_clock();
  
  
  
  ////////////////////////////////////////
  // allocate candidates memory on host //
  ////////////////////////////////////////
  // use: n_time_bins, n_hits
  // malloc: host_max_number_of_pmts_in_time_bin, host_vertex_with_max_n_pmts
  if( use_timing )
    start_cuda_clock();
  allocate_candidates_memory_on_host();
  if( use_timing )
    elapsed_memory_candidates_host += stop_cuda_clock();
  
  
  ////////////////////////////////////////////////
  // set number of blocks and threads per block //
  ////////////////////////////////////////////////
  // set: number_of_kernel_blocks, number_of_threads_per_block
  // use: n_test_vertices, n_hits
  if( use_timing )
    start_c_clock();
  if( !setup_threads_for_tof_2d() ) return 0;
  if( use_timing )
    elapsed_threads += stop_c_clock();
  
  
  
  ///////////////////////////////////////
  // allocate correct memory on device //
  ///////////////////////////////////////
  // use: n_test_vertices, n_hits, n_time_bins
  // cudamalloc: device_ids, device_times, device_n_pmts_per_time_bin
  if( use_timing )
    start_cuda_clock();
  allocate_correct_memory_on_device();
  if( use_timing )
    elapsed_memory_dev += stop_cuda_clock();
  
  
  //////////////////////////////////////
  // copy input into device variables //
  //////////////////////////////////////
  // use: n_hits
  // memcpy: device_ids, device_times, constant_time_offset
  // texture: tex_ids, tex_times
  if( use_timing )
    start_cuda_clock();
  fill_correct_memory_on_device();
  if( use_timing )
    elapsed_copy_dev += stop_cuda_clock();
  
  
  
  ////////////////////
  // execute kernel //
  ////////////////////
  if( use_timing )
    start_cuda_clock();
  printf(" --- execute kernel \n");
  kernel_correct_times<<<number_of_kernel_blocks_3d,number_of_threads_per_block_3d>>>(device_n_pmts_per_time_bin);
  getLastCudaError("correct_kernel execution failed\n");
  if( use_timing )
    elapsed_kernel += stop_cuda_clock();
  
  
  
  //////////////////////////////////
  // setup threads for candidates //
  //////////////////////////////////
  // set: number_of_kernel_blocks, number_of_threads_per_block
  // use: n_time_bins
  if( use_timing )
    start_c_clock();
  if( !setup_threads_to_find_candidates() ) return 0;
  if( use_timing )
    elapsed_threads_candidates += stop_c_clock();
  
  
  
  //////////////////////////////////////////
  // allocate candidates memory on device //
  //////////////////////////////////////////
  // use: n_time_bins
  // cudamalloc: device_max_number_of_pmts_in_time_bin, device_vertex_with_max_n_pmts
  if( use_timing )
    start_cuda_clock();
  allocate_candidates_memory_on_device();
  if( use_timing )
    elapsed_candidates_memory_dev += stop_cuda_clock();
  
  
  
  /////////////////////////////////////
  // find candidates above threshold //
  /////////////////////////////////////
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- execute candidates kernel \n");
  kernel_find_vertex_with_max_npmts_in_timebin<<<number_of_kernel_blocks,number_of_threads_per_block>>>(device_n_pmts_per_time_bin, device_max_number_of_pmts_in_time_bin, device_vertex_with_max_n_pmts);
  getLastCudaError("candidates_kernel execution failed\n");
  if( use_timing )
    elapsed_candidates_kernel += stop_cuda_clock();
  
  
  
  
  /////////////////////////////////////////
  // copy candidates from device to host //
  /////////////////////////////////////////
  // use: n_time_bins
  // memcpy: host_max_number_of_pmts_in_time_bin, host_vertex_with_max_n_pmts
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- copy candidates from device to host \n");
  copy_candidates_from_device_to_host();
  if( use_timing )
    elapsed_candidates_copy_host += stop_cuda_clock();
  

  
  ///////////////////////////////////////
  // choose candidates above threshold //
  ///////////////////////////////////////
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- choose candidates above threshold \n");
  choose_candidates_above_threshold();
  if( use_timing )
    choose_candidates = stop_cuda_clock();
  
  
  
  ///////////////////////
  // coalesce triggers //
  ///////////////////////
  if( use_timing )
    start_cuda_clock();
  coalesce_triggers();
  if( use_timing )
    elapsed_coalesce += stop_cuda_clock();
  
  
  
  
  //////////////////////////////////
  // separate triggers into gates //
  //////////////////////////////////
  if( use_timing )
    start_cuda_clock();
  separate_triggers_into_gates();
  if( use_timing )
    elapsed_gates += stop_cuda_clock();
  

  int the_output = trigger_pair_vertex_time.size(); 
  
  
  /////////////////////////////
  // deallocate event memory //
  /////////////////////////////
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- deallocate memory \n");
  free_event_memories();
  if( use_timing )
    elapsed_free += stop_cuda_clock();


  printf(" ------ analyzed event \n");

  return the_output;
}



//
// kernel routine
// 

// __global__ identifier says it's a kernel function
__global__ void kernel_correct_times(unsigned int *ct){

  // get unique id for each thread in each block
  unsigned int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
  unsigned int tid_y = threadIdx.y + blockDim.y*blockIdx.y;

  // map the two 2D indices to a single linear, 1D index
  int tid = tid_y * gridDim.x * blockDim.x + tid_x;

  // tid runs from 0 to n_test_vertices * n_hits:
  //      vertex 0           vertex 1       ...     vertex m
  // (hit 0, ..., hit n; hit 0, ..., hit n; ...; hit 0, ..., hit n);

  unsigned int vertex_index = (int)(tid/constant_n_hits);
  unsigned int hit_index = tid % constant_n_hits;

  //  printf(" tid %d tidx %d tidy %d v %d h %d \n", tid, tid_x, tid_y, vertex_index, hit_index);

  //    printf( " threadi %d blockdim %d blockid %d, tid %d, vertex_index %d, hit %d \n",
  //  	  threadIdx.x, blockDim.x, blockIdx.x, tid,
  //	  vertex_index, hit_index);

  // skip if thread is assigned to nonexistent vertex
  if( vertex_index >= constant_n_test_vertices ) return;

  // skip if thread is assigned to nonexistent hit
  if( hit_index >= constant_n_hits ) return;

  unsigned int vertex_block = constant_n_time_bins*vertex_index;

  unsigned int vertex_block2 = constant_n_PMTs*vertex_index;

  atomicAdd(&
	    ct[
	       device_get_time_index(
				     int(floor(
					       (tex1Dfetch(tex_times,hit_index)
						- tex1Dfetch(tex_times_of_flight,
							     device_get_distance_index(
										       tex1Dfetch(tex_ids,hit_index),
										       vertex_block2
										       )
							     )
						+ constant_time_offset)/constant_time_step_size
					       )
					 ),
				     vertex_block
				     )
	       ]
	    ,1);

  //  printf( " hit %d (nh %d) id %d t %d; vertex %d (nv %d) tof %f  %d \n", hit_index, constant_n_hits, ids[hit_index], t[hit_index], vertex_index, constant_n_test_vertices, tof, ct[time_index]);

  return;

}




bool read_input(std::vector<int> PMTids, std::vector<int> times, int * max_time){

  int time;
  int min = INT_MAX;
  int max = INT_MIN;
  for( unsigned int i=0; i<n_hits; i++){
    time = times[i];
    host_times[i] = times[i];
    host_ids[i] = PMTids[i];
    if( time > max ) max = time;
    if( time < min ) min = time;
  }

  if( min < 0 ){
    time_offset -= min;
  }


  *max_time = max;

  return true;

}


bool read_detector(){

  FILE *f=fopen(detector_file.c_str(), "r");
  double pmt_radius;
  if( fscanf(f, "%lf %lf %lf", &detector_height, &detector_radius, &pmt_radius) != 3 ){
    printf(" problem scanning detector \n");
    fclose(f);
    return false;
  }

  fclose(f);
  return true;

}



void print_parameters(){

  printf(" n_test_vertices = %d \n", n_test_vertices);
  printf(" n_water_like_test_vertices = %d \n", n_water_like_test_vertices);
  printf(" n_PMTs = %d \n", n_PMTs);
  printf(" number_of_kernel_blocks = %d \n", number_of_kernel_blocks);
  printf(" number_of_threads_per_block = %d \n", number_of_threads_per_block);
  printf(" grid size = %d -> %d \n", number_of_kernel_blocks*number_of_threads_per_block, grid_size);

}

void print_parameters_2d(){

  printf(" n_test_vertices = %d \n", n_test_vertices);
  printf(" n_water_like_test_vertices = %d \n", n_water_like_test_vertices);
  printf(" n_PMTs = %d \n", n_PMTs);
  printf(" number_of_kernel_blocks = (%d, %d) = %d \n", number_of_kernel_blocks_3d.x, number_of_kernel_blocks_3d.y, number_of_kernel_blocks_3d.x * number_of_kernel_blocks_3d.y);
  printf(" number_of_threads_per_block = (%d, %d) = %d \n", number_of_threads_per_block_3d.x, number_of_threads_per_block_3d.y, number_of_threads_per_block_3d.x * number_of_threads_per_block_3d.y);
  printf(" grid size = %d -> %d \n", number_of_kernel_blocks_3d.x*number_of_kernel_blocks_3d.y*number_of_threads_per_block_3d.x*number_of_threads_per_block_3d.y, grid_size);

}

void print_input(){

  for(unsigned int i=0; i<n_hits; i++)
    printf(" hit %d time %d id %d \n", i, host_times[i], host_ids[i]);

}

void print_pmts(){

  for(unsigned int i=0; i<n_PMTs; i++)
    printf(" pmt %d x %f y %f z %f  \n", i, PMT_x[i], PMT_y[i], PMT_z[i]);

}

void print_times_of_flight(){

  printf(" times_of_flight: (vertex, PMT) \n");
  unsigned int distance_index;
  for(unsigned int iv=0; iv<n_test_vertices; iv++){
    printf(" ( ");
    for(unsigned int ip=0; ip<n_PMTs; ip++){
      distance_index = get_distance_index(host_ids[ip], n_PMTs*iv);
      printf(" %f ", host_times_of_flight[distance_index]);
    }
    printf(" ) \n");
  }
}


bool read_the_pmts(){

  printf(" --- read pmts \n");
  n_PMTs = read_number_of_pmts();
  if( !n_PMTs ) return false;
  printf(" detector contains %d PMTs \n", n_PMTs);
  PMT_x = (double *)malloc(n_PMTs*sizeof(double));
  PMT_y = (double *)malloc(n_PMTs*sizeof(double));
  PMT_z = (double *)malloc(n_PMTs*sizeof(double));
  if( !read_pmts() ) return false;
  //print_pmts();
  return true;

}

bool read_the_detector(){

  printf(" --- read detector \n");
  if( !read_detector() ) return false;
  printf(" detector height %f cm, radius %f cm \n", detector_height, detector_radius);
  return true;

}

void make_test_vertices(){

  printf(" --- make test vertices \n");
  float semiheight = detector_height/2.;
  n_test_vertices = 0;
  // 1: count number of test vertices
  for(int i=-1*semiheight; i <= semiheight; i+=distance_between_vertices) {
    for(int j=-1*detector_radius; j<=detector_radius; j+=distance_between_vertices) {
      for(int k=-1*detector_radius; k<=detector_radius; k+=distance_between_vertices) {
	if(pow(j,2)+pow(k,2) > pow(detector_radius,2))
	  continue;
	n_test_vertices++;
      }
    }
  }
  vertex_x = (double *)malloc(n_test_vertices*sizeof(double));
  vertex_y = (double *)malloc(n_test_vertices*sizeof(double));
  vertex_z = (double *)malloc(n_test_vertices*sizeof(double));

  // 2: assign coordinates to test vertices
  // water-like events
  n_test_vertices = 0;
  for(int i=-1*semiheight; i <= semiheight; i+=distance_between_vertices) {
    for(int j=-1*detector_radius; j<=detector_radius; j+=distance_between_vertices) {
      for(int k=-1*detector_radius; k<=detector_radius; k+=distance_between_vertices) {

	
	if( 
	   // skip endcap region
	   abs(i) > semiheight - wall_like_distance*distance_between_vertices ||
	   // skip sidewall region
	   pow(j,2)+pow(k,2) > pow(detector_radius - wall_like_distance*distance_between_vertices,2)
	    ) continue;
	
	vertex_x[n_test_vertices] = j*1.;
	vertex_y[n_test_vertices] = k*1.;
	vertex_z[n_test_vertices] = i*1.;
	n_test_vertices++;
      }
    }
  }
  n_water_like_test_vertices = n_test_vertices;

  // wall-like events
  for(int i=-1*semiheight; i <= semiheight; i+=distance_between_vertices) {
    for(int j=-1*detector_radius; j<=detector_radius; j+=distance_between_vertices) {
      for(int k=-1*detector_radius; k<=detector_radius; k+=distance_between_vertices) {

	if( 
	   abs(i) > semiheight - wall_like_distance*distance_between_vertices ||
	   pow(j,2)+pow(k,2) > pow(detector_radius - wall_like_distance*distance_between_vertices,2)
	    ){

	  if(pow(j,2)+pow(k,2) > pow(detector_radius,2)) continue;
	  
	  vertex_x[n_test_vertices] = j*1.;
	  vertex_y[n_test_vertices] = k*1.;
	  vertex_z[n_test_vertices] = i*1.;
	  n_test_vertices++;
	}
      }
    }
  }

  return;

}

bool setup_threads_for_tof(){

  grid_size = n_test_vertices;

  number_of_kernel_blocks = grid_size / max_n_threads_per_block + 1;
  number_of_threads_per_block = ( number_of_kernel_blocks > 1 ? max_n_threads_per_block : grid_size);

  print_parameters();

  if( number_of_threads_per_block > max_n_threads_per_block ){
    printf(" warning: number_of_threads_per_block = %d cannot exceed max value %d \n", number_of_threads_per_block, max_n_threads_per_block );
    return false;
  }

  if( number_of_kernel_blocks > max_n_blocks ){
    printf(" warning: number_of_kernel_blocks = %d cannot exceed max value %d \n", number_of_kernel_blocks, max_n_blocks );
    return false;
  }

  return true;
}


bool setup_threads_for_tof_biparallel(){

  grid_size = n_test_vertices * n_hits;

  number_of_kernel_blocks = grid_size / max_n_threads_per_block + 1;
  number_of_threads_per_block = ( number_of_kernel_blocks > 1 ? max_n_threads_per_block : grid_size);

  print_parameters();

  if( number_of_threads_per_block > max_n_threads_per_block ){
    printf(" --------------------- warning: number_of_threads_per_block = %d cannot exceed max value %d \n", number_of_threads_per_block, max_n_threads_per_block );
    return false;
  }

  if( number_of_kernel_blocks > max_n_blocks ){
    printf(" warning: number_of_kernel_blocks = %d cannot exceed max value %d \n", number_of_kernel_blocks, max_n_blocks );
    return false;
  }

  return true;

}

bool setup_threads_for_tof_2d(){

  grid_size = n_test_vertices * n_hits;
  unsigned int max_n_threads_per_block_2d = sqrt(max_n_threads_per_block);

  number_of_kernel_blocks_3d.x = n_test_vertices / max_n_threads_per_block_2d + 1;
  number_of_kernel_blocks_3d.y = n_hits / max_n_threads_per_block_2d + 1;

  number_of_threads_per_block_3d.x = ( number_of_kernel_blocks_3d.x > 1 ? max_n_threads_per_block_2d : n_test_vertices);
  number_of_threads_per_block_3d.y = ( number_of_kernel_blocks_3d.y > 1 ? max_n_threads_per_block_2d : n_hits);

  print_parameters_2d();

  if( number_of_threads_per_block_3d.x > max_n_threads_per_block_2d ){
    printf(" --------------------- warning: number_of_threads_per_block x = %d cannot exceed max value %d \n", number_of_threads_per_block_3d.x, max_n_threads_per_block_2d );
    return false;
  }

  if( number_of_threads_per_block_3d.y > max_n_threads_per_block_2d ){
    printf(" --------------------- warning: number_of_threads_per_block y = %d cannot exceed max value %d \n", number_of_threads_per_block_3d.y, max_n_threads_per_block_2d );
    return false;
  }

  if( number_of_kernel_blocks_3d.x > max_n_blocks ){
    printf(" warning: number_of_kernel_blocks x = %d cannot exceed max value %d \n", number_of_kernel_blocks_3d.x, max_n_blocks );
    return false;
  }

  if( number_of_kernel_blocks_3d.y > max_n_blocks ){
    printf(" warning: number_of_kernel_blocks y = %d cannot exceed max value %d \n", number_of_kernel_blocks_3d.y, max_n_blocks );
    return false;
  }

  return true;

}

bool setup_threads_to_find_candidates(){

  number_of_kernel_blocks = n_time_bins / max_n_threads_per_block + 1;
  number_of_threads_per_block = ( number_of_kernel_blocks > 1 ? max_n_threads_per_block : n_time_bins);

  if( number_of_threads_per_block > max_n_threads_per_block ){
    printf(" warning: number_of_threads_per_block = %d cannot exceed max value %d \n", number_of_threads_per_block, max_n_threads_per_block );
    return false;
  }

  return true;
}


bool read_the_input(std::vector<int> PMTids, std::vector<int> times){

  printf(" --- read input \n");
  n_hits = PMTids.size();
  if( !n_hits ) return false;
  printf(" input contains %d hits \n", n_hits);
  host_ids = (unsigned int *)malloc(n_hits*sizeof(unsigned int));
  host_times = (unsigned int *)malloc(n_hits*sizeof(unsigned int));
  int max_time;
  if( !read_input(PMTids, times, &max_time) ) return false;
  //time_offset = 600.; // set to constant to match trevor running
  n_time_bins = int(floor((max_time + time_offset)/time_step_size))+1; // floor returns the integer below
  printf(" input max_time %d, n_time_bins %d \n", max_time, n_time_bins);
  printf(" time_offset = %f ns \n", time_offset);
  //print_input();

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constant_n_time_bins), &n_time_bins, sizeof(n_time_bins)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constant_n_hits), &n_hits, sizeof(n_hits)) );

  return true;
}

void allocate_tofs_memory_on_device(){

  printf(" --- allocate memory tofs \n");
  checkCudaErrors(hipMalloc((void **)&device_times_of_flight, n_test_vertices*n_PMTs*sizeof(float)));
  /*
    if( n_hits*n_test_vertices > available_memory ){
    printf(" cannot allocate vector of %d, available_memory %d \n", n_hits*n_test_vertices, available_memory);
    return 0;
    }
  */


  return;

}

void allocate_correct_memory_on_device(){

  printf(" --- allocate memory \n");
  /*
    if( n_hits > available_memory ){
    printf(" cannot allocate vector of %d, available_memory %d \n", n_hits, available_memory);
    return 0;
    }
  */
  checkCudaErrors(hipMalloc((void **)&device_ids, n_hits*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **)&device_times, n_hits*sizeof(unsigned int)));
  /*
    if( n_test_vertices*n_PMTs > available_memory ){
    printf(" cannot allocate vector of %d, available_memory %d \n", n_test_vertices*n_PMTs, available_memory);
    return 0;
    }
  */
  checkCudaErrors(hipMalloc((void **)&device_n_pmts_per_time_bin, n_time_bins*n_test_vertices*sizeof(unsigned int)));
  checkCudaErrors(hipMemset(device_n_pmts_per_time_bin, 0, n_time_bins*n_test_vertices*sizeof(unsigned int)));

  return;

}

void allocate_candidates_memory_on_host(){

  printf(" --- allocate candidates memory on host \n");

  host_max_number_of_pmts_in_time_bin = (unsigned int *)malloc(n_time_bins*sizeof(unsigned int));
  host_vertex_with_max_n_pmts = (unsigned int *)malloc(n_time_bins*sizeof(unsigned int));

  return;

}

void allocate_candidates_memory_on_device(){

  printf(" --- allocate candidates memory on device \n");

  checkCudaErrors(hipMalloc((void **)&device_max_number_of_pmts_in_time_bin, n_time_bins*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **)&device_vertex_with_max_n_pmts, n_time_bins*sizeof(unsigned int)));

  return;

}

void make_table_of_tofs(){

  printf(" --- fill times_of_flight \n");
  host_times_of_flight = (float*)malloc(n_test_vertices*n_PMTs * sizeof(double));
  printf(" speed_light_water %f \n", speed_light_water);
  unsigned int distance_index;
  time_offset = 0.;
  for(unsigned int ip=0; ip<n_PMTs; ip++){
    for(unsigned int iv=0; iv<n_test_vertices; iv++){
      distance_index = get_distance_index(ip + 1, n_PMTs*iv);
      host_times_of_flight[distance_index] = sqrt(pow(vertex_x[iv] - PMT_x[ip],2) + pow(vertex_y[iv] - PMT_y[ip],2) + pow(vertex_z[iv] - PMT_z[ip],2))/speed_light_water;
      if( host_times_of_flight[distance_index] > time_offset )
	time_offset = host_times_of_flight[distance_index];

    }
  }
  //print_times_of_flight();

  return;
}


void fill_tofs_memory_on_device(){

  printf(" --- copy tofs from host to device \n");
  checkCudaErrors(hipMemcpy(device_times_of_flight,
			     host_times_of_flight,
			     n_test_vertices*n_PMTs*sizeof(float),
			     hipMemcpyHostToDevice));
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constant_time_step_size), &time_step_size, sizeof(time_step_size)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constant_n_test_vertices), &n_test_vertices, sizeof(n_test_vertices)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constant_n_water_like_test_vertices), &n_water_like_test_vertices, sizeof(n_water_like_test_vertices)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constant_n_PMTs), &n_PMTs, sizeof(n_PMTs)) );

  // Bind the array to the texture
  checkCudaErrors(hipBindTexture(0,tex_times_of_flight, device_times_of_flight, n_test_vertices*n_PMTs*sizeof(float)));
  


  return;
}


void fill_correct_memory_on_device(){

  printf(" --- copy from host to device \n");
  checkCudaErrors(hipMemcpy(device_ids,
			     host_ids,
			     n_hits*sizeof(unsigned int),
			     hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(device_times,
			     host_times,
			     n_hits*sizeof(unsigned int),
			     hipMemcpyHostToDevice));
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(constant_time_offset), &time_offset, sizeof(time_offset)) );

  checkCudaErrors(hipBindTexture(0,tex_ids, device_ids, n_hits*sizeof(unsigned int)));
  checkCudaErrors(hipBindTexture(0,tex_times, device_times, n_hits*sizeof(unsigned int)));


  return;
}





unsigned int read_number_of_pmts(){

  FILE *f=fopen(pmts_file.c_str(), "r");
  if (f == NULL){
    printf(" cannot read pmts file \n");
    fclose(f);
    return 0;
  }

  unsigned int n_pmts = 0;

  for (char c = getc(f); c != EOF; c = getc(f))
    if (c == '\n')
      n_pmts ++;

  fclose(f);
  return n_pmts;

}

bool read_pmts(){

  FILE *f=fopen(pmts_file.c_str(), "r");

  double x, y, z;
  unsigned int id;
  for( unsigned int i=0; i<n_PMTs; i++){
    if( fscanf(f, "%d %lf %lf %lf", &id, &x, &y, &z) != 4 ){
      printf(" problem scanning pmt %d \n", i);
      fclose(f);
      return false;
    }
    PMT_x[id-1] = x;
    PMT_y[id-1] = y;
    PMT_z[id-1] = z;
  }

  fclose(f);
  return true;

}


void coalesce_triggers(){

  trigger_pair_vertex_time.clear();
  trigger_npmts_in_time_bin.clear();

  unsigned int vertex_index, time_lower, time_upper, number_of_pmts_in_time_bin;
  unsigned int vertex_index_prev=0, time_lower_prev=0, time_upper_prev=0, number_of_pmts_in_time_bin_prev=0;
  unsigned int max_pmt=0,max_vertex_index=0,max_time=0;
  bool first_trigger, last_trigger, coalesce_triggers;
  unsigned int trigger_index;
  for(std::vector<std::pair<unsigned int,unsigned int> >::const_iterator itrigger=candidate_trigger_pair_vertex_time.begin(); itrigger != candidate_trigger_pair_vertex_time.end(); ++itrigger){

    vertex_index =      itrigger->first;
    time_upper = itrigger->second;
    time_lower = time_upper-1;
    trigger_index = itrigger - candidate_trigger_pair_vertex_time.begin();
    number_of_pmts_in_time_bin = candidate_trigger_npmts_in_time_bin.at(trigger_index);

    first_trigger = (trigger_index == 0);
    last_trigger = (trigger_index == candidate_trigger_pair_vertex_time.size()-1);
       
    if( first_trigger ){
      if( number_of_pmts_in_time_bin > 0){
	max_pmt = number_of_pmts_in_time_bin;
	max_vertex_index = vertex_index;
	max_time = time_upper;
      }
    }
    else{
      coalesce_triggers = (std::abs((int)(max_time - time_upper)) < coalesce_time/time_step_size);

      if( coalesce_triggers ){
	if( number_of_pmts_in_time_bin > max_pmt) {
	  max_pmt = number_of_pmts_in_time_bin;
	  max_vertex_index = vertex_index;
	  max_time = time_upper;
	}
      }else{
	trigger_pair_vertex_time.push_back(std::make_pair(max_vertex_index,max_time));
	trigger_npmts_in_time_bin.push_back(max_pmt);
	max_pmt = number_of_pmts_in_time_bin;
	max_vertex_index = vertex_index;
	max_time = time_upper;     
      }
    }

    if(last_trigger){
      trigger_pair_vertex_time.push_back(std::make_pair(max_vertex_index,max_time));
      trigger_npmts_in_time_bin.push_back(max_pmt);
    }
     
    time_upper_prev = time_upper;
    time_lower_prev = time_lower;
    vertex_index_prev = vertex_index; 
    number_of_pmts_in_time_bin_prev = number_of_pmts_in_time_bin;
  }

  for(std::vector<std::pair<unsigned int,unsigned int> >::const_iterator itrigger=trigger_pair_vertex_time.begin(); itrigger != trigger_pair_vertex_time.end(); ++itrigger)
    printf(" coalesced trigger timebin %d vertex index %d \n", itrigger->first, itrigger->second);

  return;

}


void separate_triggers_into_gates(){

  final_trigger_pair_vertex_time.clear();
  unsigned int trigger_index;

  unsigned int time_start=0;
  for(std::vector<std::pair<unsigned int,unsigned int> >::const_iterator itrigger=trigger_pair_vertex_time.begin(); itrigger != trigger_pair_vertex_time.end(); ++itrigger){
    //once a trigger is found, we must jump in the future before searching for the next
    if(itrigger->second > time_start) {
      unsigned int triggertime = itrigger->second*time_step_size - time_offset;
      final_trigger_pair_vertex_time.push_back(std::make_pair(itrigger->first,triggertime));
      time_start = triggertime + trigger_gate_up;
      trigger_index = itrigger - trigger_pair_vertex_time.begin();
      output_trigger_information.clear();
      output_trigger_information.push_back(vertex_x[itrigger->first]);
      output_trigger_information.push_back(vertex_y[itrigger->first]);
      output_trigger_information.push_back(vertex_z[itrigger->first]);
      output_trigger_information.push_back(trigger_npmts_in_time_bin.at(trigger_index));
      output_trigger_information.push_back(triggertime);

      printf(" triggertime: %d, npmts: %d, x: %f, y: %f, z: %f \n", triggertime, trigger_npmts_in_time_bin.at(trigger_index), vertex_x[itrigger->first], vertex_y[itrigger->first], vertex_z[itrigger->first]);


    }
  }


  return;
}


float timedifference_msec(struct timeval t0, struct timeval t1){
  return (t1.tv_sec - t0.tv_sec) * 1000.0f + (t1.tv_usec - t0.tv_usec) / 1000.0f;
}



void start_c_clock(){
  gettimeofday(&t0,0);

}
double stop_c_clock(){
  gettimeofday(&t1,0);
  return timedifference_msec(t0, t1);
}
void start_cuda_clock(){
  hipEventRecord(start);

}
double stop_cuda_clock(){
  float milli;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  return milli;
}
void start_total_cuda_clock(){
  hipEventRecord(total_start);

}
double stop_total_cuda_clock(){
  float milli;
  hipEventRecord(total_stop);
  hipEventSynchronize(total_stop);
  hipEventElapsedTime(&milli, total_start, total_stop);
  return milli;
}

unsigned int get_distance_index(unsigned int pmt_id, unsigned int vertex_block){
  // block = (npmts) * (vertex index)

  return pmt_id - 1 + vertex_block;

}

unsigned int get_time_index(unsigned int hit_index, unsigned int vertex_block){
  // block = (n time bins) * (vertex index)

  return hit_index + vertex_block;

}

__device__ unsigned int device_get_distance_index(unsigned int pmt_id, unsigned int vertex_block){
  // block = (npmts) * (vertex index)

  return pmt_id - 1 + vertex_block;

}

__device__ unsigned int device_get_time_index(unsigned int hit_index, unsigned int vertex_block){
  // block = (n time bins) * (vertex index)

  return hit_index + vertex_block;

}

// Print device properties
void print_gpu_properties(){

  int devCount;
  hipGetDeviceCount(&devCount);
  printf(" CUDA Device Query...\n");
  printf(" There are %d CUDA devices.\n", devCount);
  hipDeviceProp_t devProp;
  for (int i = 0; i < devCount; ++i){
    // Get device properties
    printf(" CUDA Device #%d\n", i);
    hipGetDeviceProperties(&devProp, i);
    printf("Major revision number:          %d\n",  devProp.major);
    printf("Minor revision number:          %d\n",  devProp.minor);
    printf("Name:                           %s\n",  devProp.name);
    printf("Total global memory:            %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block:  %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:      %d\n",  devProp.regsPerBlock);
    printf("Warp size:                      %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:           %lu\n",  devProp.memPitch);
    max_n_threads_per_block = devProp.maxThreadsPerBlock;
    printf("Maximum threads per block:      %d\n",  max_n_threads_per_block);
    for (int i = 0; i < 3; ++i)
      printf("Maximum dimension %d of block:   %d\n", i, devProp.maxThreadsDim[i]);
    max_n_blocks = devProp.maxGridSize[0];
    for (int i = 0; i < 3; ++i)
      printf("Maximum dimension %d of grid:    %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                     %d\n",  devProp.clockRate);
    printf("Total constant memory:          %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:              %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution:  %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:      %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:       %s\n",  (devProp.kernelExecTimeoutEnabled ?"Yes" : "No"));
    printf("Memory Clock Rate (KHz):        %d\n", devProp.memoryClockRate);
    printf("Memory Bus Width (bits):        %d\n", devProp.memoryBusWidth);
    printf("Peak Memory Bandwidth (GB/s):   %f\n", 2.0*devProp.memoryClockRate*(devProp.memoryBusWidth/8)/1.0e6);
    printf("Concurrent kernels:             %d\n",  devProp.concurrentKernels);
  }
  size_t available_memory, total_memory;
  hipMemGetInfo(&available_memory, &total_memory);
  size_t stack_memory;
  hipDeviceGetLimit(&stack_memory, hipLimitStackSize);
  size_t fifo_memory;
  hipDeviceGetLimit(&fifo_memory, hipLimitPrintfFifoSize);
  size_t heap_memory;
  hipDeviceGetLimit(&heap_memory, hipLimitMallocHeapSize);
  printf(" memgetinfo: available_memory %f MB, total_memory %f MB, stack_memory %f MB, fifo_memory %f MB, heap_memory %f MB \n", (double)available_memory/1.e6, (double)total_memory/1.e6, (double)stack_memory/1.e6, (double)fifo_memory/1.e6, (double)heap_memory/1.e6);


  return;
}


__global__ void kernel_find_vertex_with_max_npmts_in_timebin(unsigned int * np, unsigned int * mnp, unsigned int * vmnp){


  // get unique id for each thread in each block == time bin
  unsigned int time_bin_index = threadIdx.x + blockDim.x*blockIdx.x;

  // skip if thread is assigned to nonexistent time bin
  if( time_bin_index >= constant_n_time_bins ) return;


  unsigned int number_of_pmts_in_time_bin = 0;
  unsigned int time_index;
  unsigned int max_number_of_pmts_in_time_bin=0;
  unsigned int vertex_with_max_n_pmts = 0;

  for(unsigned int iv=0;iv<constant_n_test_vertices;iv++) { // loop over test vertices
    // sum the number of hit PMTs in this time window
    
    time_index = time_bin_index + constant_n_time_bins*iv;
    if( time_index >= constant_n_time_bins*constant_n_test_vertices - 1 ) continue;
    number_of_pmts_in_time_bin = np[time_index] + np[time_index+1];
    if( number_of_pmts_in_time_bin > max_number_of_pmts_in_time_bin ){
      max_number_of_pmts_in_time_bin = number_of_pmts_in_time_bin;
      vertex_with_max_n_pmts = iv;
    }
  }

  mnp[time_bin_index] = max_number_of_pmts_in_time_bin;
  vmnp[time_bin_index] = vertex_with_max_n_pmts;

  return;

}

void free_event_memories(){

  checkCudaErrors(hipUnbindTexture(tex_ids));
  checkCudaErrors(hipUnbindTexture(tex_times));
  free(host_ids);
  free(host_times);
  checkCudaErrors(hipFree(device_ids));
  checkCudaErrors(hipFree(device_times));
  checkCudaErrors(hipFree(device_n_pmts_per_time_bin));
  free(host_max_number_of_pmts_in_time_bin);
  free(host_vertex_with_max_n_pmts);
  checkCudaErrors(hipFree(device_max_number_of_pmts_in_time_bin));
  checkCudaErrors(hipFree(device_vertex_with_max_n_pmts));

  return;
}

void free_global_memories(){

  //unbind texture reference to free resource 
  checkCudaErrors(hipUnbindTexture(tex_times_of_flight));

  free(PMT_x);
  free(PMT_y);
  free(PMT_z);
  free(vertex_x);
  free(vertex_y);
  free(vertex_z);
  checkCudaErrors(hipFree(device_times_of_flight));
  free(host_times_of_flight);

  return;
}

void copy_candidates_from_device_to_host(){

  checkCudaErrors(hipMemcpy(host_max_number_of_pmts_in_time_bin,
			     device_max_number_of_pmts_in_time_bin,
			     n_time_bins*sizeof(unsigned int),
			     hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(host_vertex_with_max_n_pmts,
			     device_vertex_with_max_n_pmts,
			     n_time_bins*sizeof(unsigned int),
			     hipMemcpyDeviceToHost));


}


void choose_candidates_above_threshold(){

  candidate_trigger_pair_vertex_time.clear();
  candidate_trigger_npmts_in_time_bin.clear();

  unsigned int the_threshold;

  for(unsigned int time_bin = 0; time_bin<n_time_bins - 1; time_bin++){ // loop over time bins
    // n_time_bins - 1 as we are checking the i and i+1 at the same time
    
    if(host_vertex_with_max_n_pmts[time_bin] < n_water_like_test_vertices )
      the_threshold = water_like_threshold_number_of_pmts;
    else
      the_threshold = wall_like_threshold_number_of_pmts;

    if(host_max_number_of_pmts_in_time_bin[time_bin] > the_threshold) {

      if( use_verbose )
	printf(" time %f vertex (%f, %f, %f) npmts %d \n", (time_bin + 2)*time_step_size - time_offset, vertex_x[host_vertex_with_max_n_pmts[time_bin]], vertex_y[host_vertex_with_max_n_pmts[time_bin]], vertex_z[host_vertex_with_max_n_pmts[time_bin]], host_max_number_of_pmts_in_time_bin[time_bin]);

      candidate_trigger_pair_vertex_time.push_back(std::make_pair(host_vertex_with_max_n_pmts[time_bin],time_bin+2));
      candidate_trigger_npmts_in_time_bin.push_back(host_max_number_of_pmts_in_time_bin[time_bin]);
    }

  }

  if( use_verbose )
    printf(" n candidates: %d \n", candidate_trigger_pair_vertex_time.size());
}





float read_value_from_file(std::string paramname, std::string filename){

  FILE * pFile = fopen (filename.c_str(),"r");

  char name[256];
  float value;

  while( EOF != fscanf(pFile, "%s %e", name, &value) ){
    if( paramname.compare(name) == 0 ){
      fclose(pFile);
      return value;
    }
  }

  printf(" warning: could not find parameter %s in file %s \n", paramname.c_str(), filename.c_str());

  fclose(pFile);
  exit(0);

  return 0.;

}

void read_user_parameters(std::string parameter_file){

  speed_light_water = 29.9792/1.3330; // speed of light in water, cm/ns
  //double speed_light_water = 22.490023;

  double dark_rate = read_value_from_file("dark_rate", parameter_file); // Hz
  distance_between_vertices = read_value_from_file("distance_between_vertices", parameter_file); // cm
  wall_like_distance = read_value_from_file("wall_like_distance", parameter_file); // units of distance between vertices
  time_step_size = (unsigned int)(sqrt(3.)*distance_between_vertices/(4.*speed_light_water)); // ns
  int extra_threshold = (int)(dark_rate*n_PMTs*2.*time_step_size*1.e-9); // to account for dark current occupancy
  water_like_threshold_number_of_pmts = read_value_from_file("water_like_threshold_number_of_pmts", parameter_file) + extra_threshold;
  wall_like_threshold_number_of_pmts = read_value_from_file("wall_like_threshold_number_of_pmts", parameter_file) + extra_threshold;
  coalesce_time = read_value_from_file("coalesce_time", parameter_file); // ns
  trigger_gate_up = read_value_from_file("trigger_gate_up", parameter_file); // ns
  trigger_gate_down = read_value_from_file("trigger_gate_down", parameter_file); // ns


}


int gpu_daq_initialize(std::string the_pmts_file,  std::string the_detector_file, std::string parameter_file){

  int argc = 0;
  const char* n_argv[] = {};
  const char **argv = n_argv;

  /////////////////////
  // initialise card //
  /////////////////////
  findCudaDevice(argc, argv);

  // initialise CUDA timing
  use_timing = true;
  if( use_timing ){
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }
  hipEventCreate(&total_start);
  hipEventCreate(&total_stop);
  elapsed_parameters = 0; elapsed_pmts = 0; elapsed_detector = 0; elapsed_vertices = 0;
  elapsed_threads = 0; elapsed_tof = 0; elapsed_memory_tofs_dev = 0; elapsed_memory_candidates_host = 0; elapsed_tofs_copy_dev = 0;
  elapsed_input = 0; elapsed_memory_dev = 0; elapsed_copy_dev = 0; elapsed_kernel = 0; 
  elapsed_threads_candidates = 0; elapsed_candidates_memory_dev = 0; elapsed_candidates_kernel = 0;
  elapsed_candidates_copy_host = 0; choose_candidates = 0; elapsed_coalesce = 0; elapsed_gates = 0; elapsed_free = 0; elapsed_total = 0;
  elapsed_tofs_free = 0; elapsed_reset = 0;
  use_verbose = true;


  ////////////////////
  // inspect device //
  ////////////////////
  // set: max_n_threads_per_block, max_n_blocks
  print_gpu_properties();



  ////////////////
  // read PMTs  //
  ////////////////
  // set: n_PMTs, PMT_x, PMT_y, PMT_z
  if( use_timing )
    start_c_clock();
  detector_file = the_detector_file;
  pmts_file = the_pmts_file;
  if( !read_the_pmts() ) return 0;
  if( use_timing )
    elapsed_pmts = stop_c_clock();


  ///////////////////////
  // define parameters //
  ///////////////////////
  if( use_timing )
    start_c_clock();
  read_user_parameters(parameter_file);
  if( use_verbose ){
    printf(" --- user parameters \n");
    printf(" distance between test vertices = %f cm \n", distance_between_vertices);
    printf(" time step size = %d ns \n", time_step_size);
    printf(" water_like_threshold_number_of_pmts = %d \n", water_like_threshold_number_of_pmts);
    printf(" coalesce_time = %f ns \n", coalesce_time);
    printf(" trigger_gate_up = %f ns \n", trigger_gate_up);
    printf(" trigger_gate_down = %f ns \n", trigger_gate_down);
  }
  if( use_timing )
    elapsed_parameters = stop_c_clock();




  /////////////////////
  // read detector ////
  /////////////////////
  // set: detector_height, detector_radius, pmt_radius
  if( use_timing )
    start_c_clock();
  if( !read_the_detector() ) return 0;
  if( use_timing )
    elapsed_detector = stop_c_clock();




  ////////////////////////
  // make test vertices //
  ////////////////////////
  // set: n_test_vertices, n_water_like_test_vertices, vertex_x, vertex_y, vertex_z
  // use: detector_height, detector_radius
  if( use_timing )
    start_c_clock();
  make_test_vertices();
  if( use_timing )
    elapsed_vertices = stop_c_clock();



  //////////////////////////////
  // table of times_of_flight //
  //////////////////////////////
  // set: host_times_of_flight, time_offset
  // use: n_test_vertices, vertex_x, vertex_y, vertex_z, n_PMTs, PMT_x, PMT_y, PMT_z
  // malloc: host_times_of_flight
  if( use_timing )
    start_c_clock();
  make_table_of_tofs();
  if( use_timing )
    elapsed_tof = stop_c_clock();



  ////////////////////////////////////
  // allocate tofs memory on device //
  ////////////////////////////////////
  // use: n_test_vertices, n_PMTs
  // cudamalloc: device_times_of_flight
  if( use_timing )
    start_cuda_clock();
  allocate_tofs_memory_on_device();
  if( use_timing )
    elapsed_memory_tofs_dev = stop_cuda_clock();


  ////////////////////////////////
  // fill tofs memory on device //
  ////////////////////////////////
  // use: n_test_vertices, n_water_like_test_vertices, n_PMTs
  // memcpy: device_times_of_flight, constant_time_step_size, constant_n_test_vertices, constant_n_water_like_test_vertices, constant_n_PMTs
  // texture: tex_times_of_flight
  if( use_timing )
    start_cuda_clock();
  fill_tofs_memory_on_device();
  if( use_timing )
    elapsed_tofs_copy_dev = stop_cuda_clock();




  start_total_cuda_clock();

  return 1;

}


int gpu_daq_finalize(){


  elapsed_total += stop_total_cuda_clock();


  //////////////////////////////
  // deallocate global memory //
  //////////////////////////////
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- deallocate tofs memory \n");
  free_global_memories();
  if( use_timing )
    elapsed_tofs_free = stop_cuda_clock();



  //////////////////
  // reset device //
  //////////////////
  // -- needed to flush the buffer which holds printf from each thread
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- reset device \n");
  //  hipDeviceReset();
  if( use_timing )
    elapsed_reset = stop_cuda_clock();



  //////////////////
  // print timing //
  //////////////////
  if( use_timing ){
    printf(" user parameters time : %f ms \n", elapsed_parameters);
    printf(" read pmts execution time : %f ms \n", elapsed_pmts);
    printf(" read detector execution time : %f ms \n", elapsed_detector);
    printf(" make test vertices execution time : %f ms \n", elapsed_vertices);
    printf(" setup threads candidates execution time : %f ms \n", elapsed_threads_candidates);
    printf(" make table of tofs execution time : %f ms \n", elapsed_tof);
    printf(" allocate tofs memory on device execution time : %f ms \n", elapsed_memory_tofs_dev);
    printf(" fill tofs memory on device execution time : %f ms \n", elapsed_tofs_copy_dev);
    printf(" deallocate tofs memory execution time : %f ms \n", elapsed_tofs_free);
    printf(" device reset execution time : %f ms \n", elapsed_reset);
    printf(" read input execution time : %f ms (%f) \n", elapsed_input, elapsed_input/elapsed_total);
    printf(" allocate candidates memory on host execution time : %f ms (%f) \n", elapsed_memory_candidates_host, elapsed_memory_candidates_host/elapsed_total);
    printf(" setup threads execution time : %f ms (%f) \n", elapsed_threads, elapsed_threads/elapsed_total);
    printf(" allocate memory on device execution time : %f ms (%f) \n", elapsed_memory_dev, elapsed_memory_dev/elapsed_total);
    printf(" fill memory on device execution time : %f ms (%f) \n", elapsed_copy_dev, elapsed_copy_dev/elapsed_total);
    printf(" correct kernel execution time : %f ms (%f) \n", elapsed_kernel, elapsed_kernel/elapsed_total);
    printf(" allocate candidates memory on device execution time : %f ms (%f) \n", elapsed_candidates_memory_dev, elapsed_candidates_memory_dev/elapsed_total);
    printf(" copy candidates to host execution time : %f ms (%f) \n", elapsed_candidates_copy_host, elapsed_candidates_copy_host/elapsed_total);
    printf(" choose candidates execution time : %f ms (%f) \n", choose_candidates, choose_candidates/elapsed_total);
    printf(" candidates kernel execution time : %f ms (%f) \n", elapsed_candidates_kernel, elapsed_candidates_kernel/elapsed_total);
    printf(" coalesce triggers execution time : %f ms (%f) \n", elapsed_coalesce, elapsed_coalesce/elapsed_total);
    printf(" separate triggers into gates execution time : %f ms (%f) \n", elapsed_gates, elapsed_gates/elapsed_total);
    printf(" deallocate memory execution time : %f ms (%f) \n", elapsed_free, elapsed_free/elapsed_total);
  }
  printf(" total execution time : %f ms \n", elapsed_total);

  return 1;

}
